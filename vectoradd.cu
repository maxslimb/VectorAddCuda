#include<stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include <cassert>



__global__ void vectorAdd(int *a, int * b, int *c, int n){
	//Calculate global thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid < n){
		//Each thread adds a single element
		c[tid] = a[tid] + b[tid];
	}


}

__global__ void vectorAdd2(int *A, int *B, int *C, int n){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < n){
	
		//add easch element throught row

		for(int i= 0; i<n; i++){
		
			C[tid * n + i] = B[tid * n + i] + A[tid * n + i];
		}
	
	
	}


}

void result_check(int *a, int *b, int *c, int n){
	for(int i=0;i<n;i++){
	assert(c[i]==(a[i] + b[i]));
	}

}

int  main(){

// initialize arrays
	int *a, *b, *c;
	int n=100, size =n * sizeof(int);

//allocate memory
	a = (int *) malloc(n * sizeof(int));
	b = (int *) malloc(n*sizeof(int));
	c= (int *) malloc(n*sizeof(int));
	
// initialize values
	for(int i=0;i<n;i++)
	{	
		a[i]= rand()%100;
		b[i]=rand()%100;
		c[i]=0;
//	printf("%d ",a[i]);
	}

// intialize device arrays
	int * d_a, *d_b, *d_c;
	
// allocate device memory
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

//copy data to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//Number of Threads
	int numberOfThreads = 10;

//Number of Block size
	int numberblocks = n/numberOfThreads;
	
//call device function
	vectorAdd2<<<numberblocks, numberOfThreads>>>(d_a, d_b, d_c, n);

//copy data to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

//check the result
	result_check(a, b, c, n);

	printf("Completed!");
//free all the memory
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

return 0;
}

